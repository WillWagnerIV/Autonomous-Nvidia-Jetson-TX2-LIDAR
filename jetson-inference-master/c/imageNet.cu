#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include "cudaUtility.h"


// gpuPreImageNetRGB
__global__ void gpuPreImageNetRGB( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.x, px.y, px.z);
	
	output[n * 0 + m] = bgr.x;
	output[n * 1 + m] = bgr.y;
	output[n * 2 + m] = bgr.z;
}


// cudaPreImageNetRGB
hipError_t cudaPreImageNetRGB( float4* input, size_t inputWidth, size_t inputHeight,
				            float* output, size_t outputWidth, size_t outputHeight,
					       hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetRGB<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}


// gpuPreImageNetBGR
__global__ void gpuPreImageNetBGR( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z, px.y, px.x);
	
	output[n * 0 + m] = bgr.x;
	output[n * 1 + m] = bgr.y;
	output[n * 2 + m] = bgr.z;
}


// cudaPreImageNetBGR
hipError_t cudaPreImageNetBGR( float4* input, size_t inputWidth, size_t inputHeight,
				            float* output, size_t outputWidth, size_t outputHeight,
					       hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetBGR<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}


// gpuPreImageNetMeanRGB
__global__ void gpuPreImageNetMeanRGB( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float3 mean_value )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.x - mean_value.x, px.y - mean_value.y, px.z - mean_value.z);
	
	output[n * 0 + m] = bgr.x;
	output[n * 1 + m] = bgr.y;
	output[n * 2 + m] = bgr.z;
}


// cudaPreImageNetMeanRGB
hipError_t cudaPreImageNetMeanRGB( float4* input, size_t inputWidth, size_t inputHeight,
				                float* output, size_t outputWidth, size_t outputHeight, 
						      const float3& mean_value, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetMeanRGB<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight, mean_value);

	return CUDA(hipGetLastError());
}


// gpuPreImageNetMeanBGR
__global__ void gpuPreImageNetMeanBGR( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float3 mean_value )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z - mean_value.x, px.y - mean_value.y, px.x - mean_value.z);
	
	output[n * 0 + m] = bgr.x;
	output[n * 1 + m] = bgr.y;
	output[n * 2 + m] = bgr.z;
}


// cudaPreImageNetMeanBGR
hipError_t cudaPreImageNetMeanBGR( float4* input, size_t inputWidth, size_t inputHeight,
				                float* output, size_t outputWidth, size_t outputHeight, 
						      const float3& mean_value, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetMeanBGR<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight, mean_value);

	return CUDA(hipGetLastError());
}


// gpuPreImageNetNormRGB
__global__ void gpuPreImageNetNormRGB( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float multiplier, float min_value )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.x, px.y, px.z);
	
	output[n * 0 + m] = bgr.x * multiplier + min_value;
	output[n * 1 + m] = bgr.y * multiplier + min_value;
	output[n * 2 + m] = bgr.z * multiplier + min_value;
}


// cudaPreImageNetNormRGB
hipError_t cudaPreImageNetNormRGB( float4* input, size_t inputWidth, size_t inputHeight,
							 float* output, size_t outputWidth, size_t outputHeight,
							 const float2& range, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	const float multiplier = (range.y - range.x) / 255.0f;
	
	//printf("cudaPreImageNetNorm([%f, %f])  multiplier=%f\n", range.x, range.y, multiplier);
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetNormRGB<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight, multiplier, range.x);

	return CUDA(hipGetLastError());
}


// gpuPreImageNetNormBGR
__global__ void gpuPreImageNetNormBGR( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float multiplier, float min_value )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z, px.y, px.x);
	
	output[n * 0 + m] = bgr.x * multiplier + min_value;
	output[n * 1 + m] = bgr.y * multiplier + min_value;
	output[n * 2 + m] = bgr.z * multiplier + min_value;
}


// cudaPreImageNetNorm
hipError_t cudaPreImageNetNormBGR( float4* input, size_t inputWidth, size_t inputHeight,
								 float* output, size_t outputWidth, size_t outputHeight,
								 const float2& range, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	const float multiplier = (range.y - range.x) / 255.0f;
	
	//printf("cudaPreImageNetNorm([%f, %f])  multiplier=%f\n", range.x, range.y, multiplier);
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetNormBGR<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight, multiplier, range.x);

	return CUDA(hipGetLastError());
}



// gpuPreImageNetNormMeanRGB
__global__ void gpuPreImageNetNormMeanRGB( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float multiplier, float min_value, const float3 mean, const float3 stdDev )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.x * multiplier + min_value, px.y * multiplier + min_value, px.z * multiplier + min_value);
	
	output[n * 0 + m] = (bgr.x - mean.x) / stdDev.x;
	output[n * 1 + m] = (bgr.y - mean.y) / stdDev.y;
	output[n * 2 + m] = (bgr.z - mean.z) / stdDev.z;
}


// cudaPreImageNetNormMeanRGB
hipError_t cudaPreImageNetNormMeanRGB( float4* input, size_t inputWidth, size_t inputHeight, float* output, size_t outputWidth, size_t outputHeight, const float2& range, const float3& mean, const float3& stdDev, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	const float multiplier = (range.y - range.x) / 255.0f;
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetNormMeanRGB<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight, multiplier, range.x, mean, stdDev);

	return CUDA(hipGetLastError());
}


// gpuPreImageNetNormMeanBGR
__global__ void gpuPreImageNetNormMeanBGR( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float multiplier, float min_value, const float3 mean, const float3 stdDev )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= oWidth || y >= oHeight )
		return;

	const int n = oWidth * oHeight;
	const int m = y * oWidth + x;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z * multiplier + min_value, px.y * multiplier + min_value, px.x * multiplier + min_value);
	
	output[n * 0 + m] = (bgr.x - mean.x) / stdDev.x;
	output[n * 1 + m] = (bgr.y - mean.y) / stdDev.y;
	output[n * 2 + m] = (bgr.z - mean.z) / stdDev.z;
}


// cudaPreImageNetNormMeanBGR
hipError_t cudaPreImageNetNormMeanBGR( float4* input, size_t inputWidth, size_t inputHeight, float* output, size_t outputWidth, size_t outputHeight, const float2& range, const float3& mean, const float3& stdDev, hipStream_t stream )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	const float multiplier = (range.y - range.x) / 255.0f;
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetNormMeanBGR<<<gridDim, blockDim, 0, stream>>>(scale, input, inputWidth, output, outputWidth, outputHeight, multiplier, range.x, mean, stdDev);

	return CUDA(hipGetLastError());
}




